#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <chrono>
#include <stdio.h>


void matrixMultiplyCuBLAS(const float* A, const float* B, float* C, int N) {
    float alpha = 1.0f;  // Scalar multiplier for AB
    float beta = 0.0f;   // Scalar multiplier for C

    size_t size = N * N * sizeof(float);
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy matrices from host to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,  
        N, N, N,                  
        &alpha,                   
        d_A, N,                   
        d_B, N,                   
        &beta,                    
        d_C, N                   
    );

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int N = 100;  

    float* A = new float[N * N];
    float* B = new float[N * N];
    float* C = new float[N * N];

    for (int i = 0; i < N * N; i++) {
        A[i] = 1.0f;  
        B[i] = 1.0f;
    }

    auto start = std::chrono::high_resolution_clock::now();
    matrixMultiplyCuBLAS(A, B, C, N);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Time taken " << elapsed.count() << " seconds" << std::endl;


    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}
